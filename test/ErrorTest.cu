#include "hip/hip_runtime.h"
#include "CuTileRasterizer.hpp"
#include "ErrorTest.hpp"
#include "GSModel.hpp"

#include <array>
#include <filesystem>
#include <fstream>
#include <nlohmann/json.hpp>
#include <random>
#include <stb_image_write.h>
#include <vector>

#define cudaCheckError() \
	{ \
		hipError_t e = hipGetLastError(); \
		if (e != hipSuccess) { \
			printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
			exit(0); \
		} \
	}

namespace cuperftest {
void WritePixelsPNG(const std::filesystem::path &filename, const float *devicePixels, uint32_t width, uint32_t height) {
	uint32_t pixelCount = width * height;
	std::vector<float> pixels(3 * pixelCount);
	hipMemcpy(pixels.data(), devicePixels, pixels.size() * sizeof(float), hipMemcpyDeviceToHost);
	cudaCheckError();

	const auto float2byte = [](float x) {
		x = x * 255.0f + 0.5f;
		auto u = (int)std::trunc(x);
		u = std::min(std::max(u, 0), 255);
		return (uint8_t)u;
	};

	std::vector<std::array<uint8_t, 3>> bytes(pixelCount);
	for (uint32_t i = 0; i < pixelCount; ++i) {
		bytes[i] = {
		    float2byte(pixels[i]),
		    float2byte(pixels[i + pixelCount]),
		    float2byte(pixels[i + pixelCount * 2]),
		};
	}

	stbi_write_png(filename.string().c_str(), (int)width, (int)height, 3, bytes.data(), (int)width * 3);
}
void RandomPixels(float *devicePixels, uint32_t width, uint32_t height) {
	uint32_t pixelCount = width * height;
	std::vector<float> pixels(3 * pixelCount);
	std::mt19937 randGen{0};
	for (float &pixel : pixels) {
		pixel = std::uniform_real_distribution<float>{-1.0f, 1.0f}(randGen);
	}
	hipMemcpy(devicePixels, pixels.data(), pixels.size() * sizeof(float), hipMemcpyHostToDevice);
	cudaCheckError();
}
void ClearDL_DSplats(const CuTileRasterizer::SplatArgs &splats, uint32_t splatCount) {
	hipMemset(splats.means, 0, splatCount * sizeof(GSModel::Mean));
	hipMemset(splats.scales, 0, splatCount * sizeof(GSModel::Scale));
	hipMemset(splats.rotates, 0, splatCount * sizeof(GSModel::Rotate));
	hipMemset(splats.opacities, 0, splatCount * sizeof(GSModel::Opacity));
	hipMemset(splats.shs, 0, splatCount * sizeof(GSModel::SH));
}
void WriteDL_DSplatsJSON(const std::filesystem::path &filename, const CuTileRasterizer::SplatArgs &splats,
                         uint32_t splatCount) {
	std::vector<GSModel::Mean> dL_dMeans(splatCount);
	std::vector<GSModel::Scale> dL_dScales(splatCount);
	std::vector<GSModel::Opacity> dL_dOpacities(splatCount);
	std::vector<GSModel::Rotate> dL_dRotates(splatCount);

	hipMemcpy(dL_dMeans.data(), splats.means, splatCount * sizeof(GSModel::Mean), hipMemcpyDeviceToHost);
	hipMemcpy(dL_dScales.data(), splats.scales, splatCount * sizeof(GSModel::Scale), hipMemcpyDeviceToHost);
	hipMemcpy(dL_dOpacities.data(), splats.opacities, splatCount * sizeof(GSModel::Opacity), hipMemcpyDeviceToHost);
	hipMemcpy(dL_dRotates.data(), splats.rotates, splatCount * sizeof(GSModel::Rotate), hipMemcpyDeviceToHost);

	nlohmann::json json;
	json["dL_dMeans"] = dL_dMeans;
	json["dL_dScales"] = dL_dScales;
	json["dL_dOpacities"] = dL_dOpacities;
	json["dL_dRotates"] = dL_dRotates;

	std::ofstream fout{filename};
	fout << json.dump(4, ' ') << std::endl;
}

myvk::Ptr<myvk::PhysicalDevice> SelectPhysicalDevice(const myvk::Ptr<myvk::Instance> &pInstance) {
	auto pPhysicalDevices = myvk::PhysicalDevice::Fetch(pInstance);
	int cuDevice{};
	hipGetDevice(&cuDevice);
	hipDeviceProp_t cuDeviceProp{};
	hipGetDeviceProperties(&cuDeviceProp, cuDevice);
	printf("CUDA Device [%d]: %s\n", cuDevice, cuDeviceProp.name);
	for (const auto &pPhysicalDevice : pPhysicalDevices) {
		if (strncmp((const char *)pPhysicalDevice->GetProperties().vk11.deviceUUID, cuDeviceProp.uuid.bytes,
					VK_UUID_SIZE) == 0)
			return pPhysicalDevice;
	}
	printf("Failed to find vkPhysicalDevice identical to CUDA Device\n");
	return nullptr;
}

} // namespace cuperftest

void GSGradient::Update(const CuTileRasterizer::SplatArgs &splats, uint32_t splatCount) {
	if (splatCount != this->splatCount) {
		this->splatCount = splatCount;
		values.resize(GetValueCount());
	}

	float *dst = values.data();
	hipMemcpy(dst, splats.means, splatCount * sizeof(Mean), hipMemcpyDeviceToHost);
	dst += splatCount * (sizeof(Mean) / sizeof(float));
	hipMemcpy(dst, splats.scales, splatCount * sizeof(Scale), hipMemcpyDeviceToHost);
	dst += splatCount * (sizeof(Scale) / sizeof(float));
	hipMemcpy(dst, splats.opacities, splatCount * sizeof(Opacity), hipMemcpyDeviceToHost);
	dst += splatCount * (sizeof(Opacity) / sizeof(float));
	hipMemcpy(dst, splats.rotates, splatCount * sizeof(Rotate), hipMemcpyDeviceToHost);
	dst += splatCount * (sizeof(Rotate) / sizeof(float));
	hipMemcpy(dst, splats.shs, splatCount * sizeof(GSModel::SH), hipMemcpyDeviceToHost);
}
