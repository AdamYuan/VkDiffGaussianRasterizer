//
// Created by adamyuan on 4/1/25.
//

#include "CuTileRasterizer.hpp"

#include "GSModel.hpp"
#include "VkCuBuffer.hpp"
#include <rasterizer.h>

#define cudaCheckError() \
	{ \
		hipError_t e = hipGetLastError(); \
		if (e != hipSuccess) { \
			printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
			exit(0); \
		} \
	}

char *CuTileRasterizer::Resource::ResizeableBuffer::Update(std::size_t updateSize) {
	if (updateSize > size) {
		if (data)
			hipFree(data);
		hipMalloc(&data, updateSize);
		cudaCheckError();
	}
	return data;
}

void CuTileRasterizer::CameraArgs::Update(const vkgsraster::Rasterizer::CameraArgs &vkCamera) {
	width = (int)vkCamera.width;
	height = (int)vkCamera.height;
	tanFovX = vkgsraster::Rasterizer::CameraArgs::GetTanFovFromFocal(vkCamera.focalX, vkCamera.width);
	tanFovY = vkgsraster::Rasterizer::CameraArgs::GetTanFovFromFocal(vkCamera.focalY, vkCamera.height);

	using Mat4 = std::array<float, 16>;
	using Mat3 = std::array<float, 9>;
	const auto $4 = [](Mat4 &mat, uint32_t i, uint32_t j) -> float & { return mat[i + j * 4]; };
	const auto $4c = [](const Mat4 &mat, uint32_t i, uint32_t j) -> float { return mat[i + j * 4]; };
	const auto $3c = [](const Mat3 &mat, uint32_t i, uint32_t j) -> float { return mat[i + j * 3]; };
	const auto mul4 = [&](const Mat4 &l, const Mat4 &r) -> Mat4 {
		Mat4 m{};
		for (uint32_t k = 0; k < 4; ++k)
			for (uint32_t j = 0; j < 4; ++j)
				for (uint32_t i = 0; i < 4; ++i) {
					$4(m, i, j) += $4c(l, i, k) * $4c(r, k, j);
				}
		return m;
	};

	Mat4 camViewMat{};
	{
		for (uint32_t j = 0; j < 3; ++j)
			for (uint32_t i = 0; i < 3; ++i)
				$4(camViewMat, i, j) = $3c(vkCamera.viewMat, i, j);
		for (uint32_t i = 0; i < 3; ++i)
			$4(camViewMat, i, 3) = -(vkCamera.pos[0] * $4c(camViewMat, i, 0)   //
			                         + vkCamera.pos[1] * $4c(camViewMat, i, 1) //
			                         + vkCamera.pos[2] * $4c(camViewMat, i, 2));
		$4(camViewMat, 3, 3) = 1.0f;
	}

	Mat4 camProjMat{};
	{
		float zNear = 0.01f;
		float zFar = 100.0f;
		float top = tanFovY * zNear;
		float bottom = -top;
		float right = tanFovX * zNear;
		float left = -right;
		float zSign = 1.0f;
		$4(camProjMat, 0, 0) = 2.0f * zNear / (right - left);
		$4(camProjMat, 1, 1) = 2.0f * zNear / (top - bottom);
		$4(camProjMat, 0, 2) = (right + left) / (right - left);
		$4(camProjMat, 1, 2) = (top + bottom) / (top - bottom);
		$4(camProjMat, 3, 2) = zSign;
		$4(camProjMat, 2, 2) = zSign * zFar / (zFar - zNear);
		$4(camProjMat, 2, 3) = -(zFar * zNear) / (zFar - zNear);

		camProjMat = mul4(camProjMat, camViewMat);
	}

	if (projMat)
		hipFree((void *)projMat);
	float *deviceCamProjMat;
	hipMalloc(&deviceCamProjMat, sizeof(Mat4));
	hipMemcpy(deviceCamProjMat, camProjMat.data(), sizeof(Mat4), hipMemcpyHostToDevice);
	projMat = deviceCamProjMat;
	cudaCheckError();

	if (viewMat)
		hipFree((void *)viewMat);
	float *deviceCamViewMat;
	hipMalloc(&deviceCamViewMat, sizeof(Mat4));
	hipMemcpy(deviceCamViewMat, camViewMat.data(), sizeof(Mat4), hipMemcpyHostToDevice);
	viewMat = deviceCamViewMat;
	cudaCheckError();

	if (pos)
		hipFree((void *)pos);
	float *deviceCamPos;
	hipMalloc(&deviceCamPos, sizeof(vkCamera.pos));
	hipMemcpy(deviceCamPos, vkCamera.pos.data(), sizeof(vkCamera.pos), hipMemcpyHostToDevice);
	pos = deviceCamPos;
	cudaCheckError();
}

void CuTileRasterizer::FwdROArgs::Update(const vkgsraster::Rasterizer::FwdROArgs &vkROArgs) {
	camera.Update(vkROArgs.camera);
	splatCount = vkROArgs.splatCount;

	splats = {
	    .means = std::static_pointer_cast<VkCuBuffer>(vkROArgs.splats.pMeanBuffer)->GetCudaMappedPtr<float>(),
	    .scales = std::static_pointer_cast<VkCuBuffer>(vkROArgs.splats.pScaleBuffer)->GetCudaMappedPtr<float>(),
	    .rotates = std::static_pointer_cast<VkCuBuffer>(vkROArgs.splats.pRotateBuffer)->GetCudaMappedPtr<float>(),
	    .opacities = std::static_pointer_cast<VkCuBuffer>(vkROArgs.splats.pOpacityBuffer)->GetCudaMappedPtr<float>(),
	    .shs = std::static_pointer_cast<VkCuBuffer>(vkROArgs.splats.pSHBuffer)->GetCudaMappedPtr<float>(),
	};

	if (bgColor)
		hipFree((void *)bgColor);
	float *deviceBgColor;
	hipMalloc(&deviceBgColor, sizeof(vkROArgs.bgColor));
	hipMemcpy(deviceBgColor, vkROArgs.bgColor.data(), sizeof(vkROArgs.bgColor), hipMemcpyHostToDevice);
	bgColor = deviceBgColor;
	cudaCheckError();
}

void CuTileRasterizer::FwdRWArgs::Update(const vkgsraster::Rasterizer::FwdRWArgs &vkRWArgs) {
	outColor = std::static_pointer_cast<VkCuBuffer>(vkRWArgs.pOutColorBuffer)->GetCudaMappedPtr<float>();
}

void CuTileRasterizer::Forward(const FwdROArgs &roArgs, const FwdRWArgs &rwArgs, Resource &resource) {
	hipDeviceSynchronize();

	// Begin
	CudaRasterizer::Rasterizer::forward(
	    [&](std::size_t size) { return resource.geometryBuffer.Update(size); },
	    [&](std::size_t size) { return resource.binningBuffer.Update(size); },
	    [&](std::size_t size) { return resource.imageBuffer.Update(size); }, (int)roArgs.splatCount, GSModel::kSHDegree,
	    GSModel::kSHSize, roArgs.bgColor, roArgs.camera.width, roArgs.camera.height, roArgs.splats.means,
	    roArgs.splats.shs, nullptr, roArgs.splats.opacities, roArgs.splats.scales, 1.0f, roArgs.splats.rotates, nullptr,
	    roArgs.camera.viewMat, roArgs.camera.projMat, roArgs.camera.pos, roArgs.camera.tanFovX, roArgs.camera.tanFovY,
	    false, rwArgs.outColor);
	hipDeviceSynchronize();
	// End
}